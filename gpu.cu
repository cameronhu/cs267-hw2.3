#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_THREADS 256
#define INDEX(row, col) ((row) * numBoxes1D + (col))

// Put any static global variables here that you will use throughout the simulation.
int blks;
double boxSize1D = cutoff;
int numBoxes1D;
int totalBoxes;
size_t boxesMemSize;
size_t prefixMemSize;

// ============ Array pointers for boxes and particle_idx ============

// CPU arrays
int* boxCounts;
int* prefixSums;
int* particle_ids;
int* boxes;

// GPU arrays
int* gpu_boxCounts;
int* gpu_prefixSums;
int* gpu_particle_ids;
int* gpu_boxes;

// =================
// Helper Functions
// =================

/**
* Helper function to calculate the box index of a given particle
*/
int findBox(const particle_t& p) {
    int col = floor(p.x / boxSize1D);
    int row = floor(p.y / boxSize1D);
    return INDEX(row, col);
}


__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;
    // r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //
    //  very simple short-range repulsive force
    //
    double coef = (1 - cutoff / r) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

__global__ void compute_forces_gpu(particle_t* particles, int num_parts) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particles[tid].ax = particles[tid].ay = 0;
    for (int j = 0; j < num_parts; j++)
        apply_force_gpu(particles[tid], particles[j]);
}

__global__ void move_gpu(particle_t* particles, int num_parts, double size) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particle_t* p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    //
    //  bounce from walls
    //
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}

void assignToBoxes(particle_t* parts, int num_parts) {
    setbuf(stdout, NULL);
    printf("Inside assignToBoxes\n");

    // Copy from parts (gpu_parts) to cpu_parts
    particle_t* cpu_parts = new particle_t[num_parts];
    hipMemcpy(cpu_parts, parts, num_parts * sizeof(particle_t), hipMemcpyDeviceToHost);
    size_t actual_size = 0;
    for (int i = 0; i < num_parts; ++i) {
        actual_size += sizeof(cpu_parts[i]);
    }
    printf("Actual size of cpu_parts: %lu\n", actual_size);
    printf("Num particles in cpu_parts: %lu\n", actual_size / sizeof(particle_t));
    
    // First pass: count particles in each box. Reset box counts from past iteration
    memset(boxCounts, 0, boxesMemSize);
    for (int i = 0; i < num_parts; ++i) {
        // printf("cur parts idx: %i\n", i);
        int boxIndex = findBox(cpu_parts[i]);
        // printf("boxIndex: %i\n", boxIndex);
        boxCounts[boxIndex]++;
    }

    // printf("Fin counting particles per box\n");

    // Compute starting index for each box in particle_idx
    int prefixSum = 0;
    for (int boxIndex = 0; boxIndex <= totalBoxes; ++boxIndex) {
        prefixSums[boxIndex] = prefixSum;
        prefixSum += boxCounts[boxIndex];
        // printf("%i\n", boxCounts[boxIndex]);
    }
    // printf("Last value of prefix sums should be num_parts. prefixSums[-1]: %i. num_parts: %i\n", prefixSums[totalBoxes], num_parts);

    // printf("Fin calc starting particle_idx index for each box's first part\n");

    // Reset box counts for use in the second pass
    memset(boxCounts, 0, boxesMemSize);

    // Second pass: assign particles to particle_idx and update boxes
    for (int i = 0; i < num_parts; ++i) {
        int boxIndex = findBox(cpu_parts[i]);
        int pos = prefixSums[boxIndex] + boxCounts[boxIndex];
        particle_ids[pos] = i;
        boxCounts[boxIndex]++;
    }
    // printf("Fin second pass to assign particles `parts` index to particle_ids in proper box order.\n");

    // Update boxes array: -1 if box has no particles
    // for (int i = 0; i < totalBoxes; ++i) {
    //     boxes[i] = (boxCounts[i] > 0) ? prefixSums[i] : -1;
    // }
    int numEmpty = 0;
    int numFilled = 0;
    int partCount = 0;
    for (int i = 0; i < totalBoxes; ++i) {
        prefixSums[i] = (boxCounts[i] > 0) ? prefixSums[i] : -1;
        if (boxCounts[i] == 0) {
            // printf("Check for empty box %i. prefixSums[%i]: %i\n", i, i, prefixSums[i]);
            numEmpty += 1;
        }
        else {
            numFilled += 1;
            partCount += boxCounts[i];
        }
    }
    // printf("Updating `boxes` array with starting indices if box has particles.\n");
    printf("prefixSums[0]: %i. prefixSums[totalBoxes]: %i\n", prefixSums[0], prefixSums[totalBoxes]);
    printf("Num empty boxes: %i. Num boxes w/ particles: %i. Num particles: %i. Average particles per box: %f\n", 
        numEmpty, numFilled, partCount, numFilled/partCount);

    // ================ Copy all CPU arrays to mirrored GPU arrays ================

}

void init_simulation(particle_t* parts, int num_parts, double size) {
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here
    setbuf(stdout, NULL);

    // Assign global variables
    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;
    numBoxes1D = ceil(size / boxSize1D);
    totalBoxes = numBoxes1D * numBoxes1D;
    boxesMemSize = totalBoxes * sizeof(int);
    prefixMemSize = (totalBoxes + 1) * sizeof(int);

    // Allocate memory for CPU-side arrays
    boxCounts = new int[totalBoxes]();
    prefixSums = new int[totalBoxes + 1];
    particle_ids = new int[num_parts];
    boxes = new int[totalBoxes];

    // Allocate memory for GPU-side arrays and copy from CPU-side arrays
    hipMalloc((void**)&gpu_boxCounts, boxesMemSize);
    hipMemset(gpu_boxCounts, 0, boxesMemSize);
    // hipMemcpy(gpu_boxCounts, boxCounts, boxesMemSize, hipMemcpyHostToDevice);

    hipMalloc((void**)&gpu_prefixSums, prefixMemSize);

    hipMalloc((void**)&gpu_particle_ids, num_parts * sizeof(int));

    hipMalloc((void**)&gpu_boxes, boxesMemSize);
    printf("Numboxes1d: %i. totalBoxes: %i\n", numBoxes1D, totalBoxes);
}

void simulate_one_step(particle_t* parts, int num_parts, double size) {
    // parts live in GPU memory
    // Rewrite this function

    // Assign all particles to boxes
    assignToBoxes(parts, num_parts);

    // Compute forces
    compute_forces_gpu<<<blks, NUM_THREADS>>>(parts, num_parts);

    // Move particles
    move_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, size);
}
