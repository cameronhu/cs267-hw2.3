#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdexcept>

#define NUM_THREADS 256
#define INDEX(row, col) ((row) * numBoxes1D + (col))

// Put any static global variables here that you will use throughout the simulation.
int blks;
double boxSize1D = cutoff;
int numBoxes1D;
int totalBoxes;
size_t boxesMemSize;
size_t prefixMemSize;
size_t particle_idMemSize;

// ============ Array pointers for boxes and particle_idx ============

// CPU arrays
int* boxCounts;
int* prefixSums;
int* particle_ids;
int* boxes;

// GPU arrays
int* gpu_boxCounts;
int* gpu_prefixSums;
int* gpu_particle_ids;
int* gpu_boxes;

// =================
// Helper Functions
// =================

/**
* Helper function to calculate the box index of a given particle
*/
int findBox(const particle_t& p) {
    int col = floor(p.x / boxSize1D);
    int row = floor(p.y / boxSize1D);
    return INDEX(row, col);
}

void getParticleIndexFromBox() {}


__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;
    // r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //
    //  very simple short-range repulsive force
    //
    double coef = (1 - cutoff / r) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

__global__ void compute_forces_gpu(particle_t* particles, int num_parts) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particles[tid].ax = particles[tid].ay = 0;
    for (int j = 0; j < num_parts; j++)
        apply_force_gpu(particles[tid], particles[j]);
}

__global__ void move_gpu(particle_t* particles, int num_parts, double size) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particle_t* p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    //
    //  bounce from walls
    //
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}

// Iterates through parts and increments boxCounts
void countParticlesPerBox(particle_t* parts, int num_parts) {
    memset(boxCounts, 0, boxesMemSize);
    for (int i = 0; i < num_parts; ++i) {
        int boxIndex = findBox(parts[i]);
        // printf("cur parts idx: %i. boxIndex: %i\n", i, boxIndex);
        boxCounts[boxIndex]++;
    }
}

// Iterates through boxCounts and computes a prefixSum
void computePrefixSum() {
    int prefixSum = 0;
    for (int boxIndex = 0; boxIndex <= totalBoxes; ++boxIndex) {
        if (boxCounts[boxIndex] > 0) {
            prefixSums[boxIndex] = prefixSum;
            prefixSum += boxCounts[boxIndex];
        } else {
            prefixSums[boxIndex] = -1;
        }
        // printf("%i\n", boxCounts[boxIndex]);
    }
}

// Organizes parts by box, in particle_id array
// Uses prefixSum and a reset boxCounts to compute where in particle_id the particle should be inserted 
void populateParticleID(particle_t* parts, int num_parts) {
    memset(boxCounts, 0, boxesMemSize);
    for (int i = 0; i < num_parts; ++i) {
        int boxIndex = findBox(parts[i]);
        if (prefixSums[boxIndex] == -1) {
            fprintf(stderr, "Populate Particle ID Error. Particle ID: %i. boxIndex: %i.\n", i, boxIndex);
            throw std::runtime_error("Populate Particle ID Error. Box found has negative prefixSum");
        }
        int pos = prefixSums[boxIndex] + boxCounts[boxIndex];
        particle_ids[pos] = i;
        boxCounts[boxIndex]++;
    }
}

void printAssignmentStats(particle_t* parts) {
    int numEmpty = 0;
    int numFilled = 0;
    int partCount = 0;
    for (int i = 0; i < totalBoxes; ++i) {
        // prefixSums[i] = (boxCounts[i] > 0) ? prefixSums[i] : -1;
        if (boxCounts[i] == 0) {
            printf("Check for empty box %i. prefixSums[%i]: %i\n", i, i, prefixSums[i]);
            numEmpty += 1;
        }
        else {
            numFilled += 1;
            partCount += boxCounts[i];
            printf("Particle id: %i with coords: (%f, %f) in box %i. findBox_output: %i.\n",
            particle_ids[prefixSums[i]], parts[particle_ids[prefixSums[i]]].x, parts[particle_ids[prefixSums[i]]].y, i, findBox(parts[particle_ids[prefixSums[i]]]));
        }
    }
    printf("prefixSums[0]: %i. prefixSums[totalBoxes]: %i\n", prefixSums[0], prefixSums[totalBoxes]);
    printf("Num empty boxes: %i. Num boxes w/ particles: %i. Num particles: %i. Average particles per box: %f\n", 
        numEmpty, numFilled, partCount, (double)(partCount / numFilled));
}

// Initializes the particle_id and prefixSums arrays, on CPU
void assignToBoxes(particle_t* parts, int num_parts) {
    // setbuf(stdout, NULL);

    // Copy from parts (gpu_parts) to cpu_parts
    particle_t* cpu_parts = new particle_t[num_parts];
    hipMemcpy(cpu_parts, parts, num_parts * sizeof(particle_t), hipMemcpyDeviceToHost);
    
    // First pass: count particles in each box. Reset box counts from past iteration
    countParticlesPerBox(cpu_parts, num_parts);

    // Compute starting index for each box in particle_idx
    computePrefixSum();

    populateParticleID(cpu_parts, num_parts);

    // printAssignmentStats(cpu_parts);
}

// Copies data from CPU particle_id and prefixSums to mirrored arrs on GPU
void copyArraysToGPU() {
    hipMemcpy(gpu_particle_ids, particle_ids, particle_idMemSize, hipMemcpyHostToDevice);
    hipMemcpy(gpu_prefixSums, prefixSums, prefixMemSize, hipMemcpyHostToDevice);
}

void init_simulation(particle_t* parts, int num_parts, double size) {
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here
    setbuf(stdout, NULL);

    // Assign global variables
    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;
    numBoxes1D = (int) ceil(size / boxSize1D);
    totalBoxes = numBoxes1D * numBoxes1D;
    boxesMemSize = totalBoxes * sizeof(int);
    prefixMemSize = (totalBoxes + 1) * sizeof(int);
    particle_idMemSize = num_parts * sizeof(particle_t);

    // Allocate memory for CPU-side arrays
    boxCounts = new int[totalBoxes]();
    prefixSums = new int[totalBoxes + 1];
    particle_ids = new int[num_parts];
    // boxes = new int[totalBoxes];

    // Allocate memory for GPU-side arrays and copy from CPU-side arrays
    hipMalloc((void**)&gpu_boxCounts, boxesMemSize);
    hipMemset(gpu_boxCounts, 0, boxesMemSize);
    // hipMemcpy(gpu_boxCounts, boxCounts, boxesMemSize, hipMemcpyHostToDevice);

    hipMalloc((void**)&gpu_prefixSums, prefixMemSize);

    hipMalloc((void**)&gpu_particle_ids, num_parts * sizeof(int));

    // hipMalloc((void**)&gpu_boxes, boxesMemSize);
    printf("Numboxes1d: %i. totalBoxes: %i\n", numBoxes1D, totalBoxes);
}

void simulate_one_step(particle_t* parts, int num_parts, double size) {
    // parts live in GPU memory
    // Rewrite this function

    // Assign all particles to boxes
    assignToBoxes(parts, num_parts);

    // Compute forces
    compute_forces_gpu<<<blks, NUM_THREADS>>>(parts, num_parts);

    // Move particles
    move_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, size);
}
