#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdexcept>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <numeric>

#define NUM_THREADS 256
#define INDEX(row, col) ((row) * numBoxes1D + (col))

// Put any static global variables here that you will use throughout the simulation.
int blks;
double boxSize1D = cutoff;
int numBoxes1D;
int totalBoxes;
size_t boxesMemSize;
size_t prefixMemSize;
size_t particle_idMemSize;

// ============ Array pointers for boxes and particle_idx ============

// CPU arrays
int* boxCounts;
int* prefixSums;
int* particle_ids;

// GPU arrays
int* gpu_boxCounts;
int* gpu_prefixSums;
int* gpu_particle_ids;

// =================
// Helper Functions
// =================

// Calculate the box row of the particle
__device__ __host__ int findRow(const particle_t& p, double boxSize1D) {
    return floor(p.y / boxSize1D);
}

// Calculate the box column of the particle
__device__ __host__ int findCol(const particle_t& p, double boxSize1D) {
    return floor(p.x / boxSize1D);
}

/**
* Helper function to calculate the box index of a given particle
*/
__device__ __host__ int findBox(const particle_t& p, int numBoxes1D, double boxSize1D) {
    int col = floor(p.x / boxSize1D);
    int row = floor(p.y / boxSize1D);
    return INDEX(row, col);
}

__device__ void apply_force_gpu(particle_t& particle, particle_t& neighbor) {
    double dx = neighbor.x - particle.x;
    double dy = neighbor.y - particle.y;
    double r2 = dx * dx + dy * dy;
    if (r2 > cutoff * cutoff)
        return;
    // r2 = fmax( r2, min_r*min_r );
    r2 = (r2 > min_r * min_r) ? r2 : min_r * min_r;
    double r = sqrt(r2);

    //
    //  very simple short-range repulsive force
    //
    double coef = (1 - cutoff / r) / r2 / mass;
    particle.ax += coef * dx;
    particle.ay += coef * dy;
}

/**
 * Given a particle, apply force from all other particles in the given box (row, col)
 * @param row Row of the neighbor box
 * @param col Column of the neighbor box
 * @param thisParticle Particle to apply force to
 */
__device__ void apply_force_from_neighbor_gpu(int row, int col, particle_t& thisParticle, particle_t* particles, int* particle_ids, int* prefixSums, int numBoxes1D, int boxSize1D) {
    // Check if the neighbor is within bounds
    if (row >= 0 && row < numBoxes1D && col >= 0 && col < numBoxes1D) {
        int boxIndex = INDEX(row, col);
        int startIdx = prefixSums[boxIndex];
        int endIdx = prefixSums[boxIndex + 1];

        // Check if there are particles in the box
        // A box with no particles will have same prefixSum as the next box
        // Apply forces for all particles in this neighboring box
        for (int i = startIdx; i < endIdx; ++i) {
            int parts_idx = particle_ids[i];
            particle_t& neighbor = particles[parts_idx];
            apply_force_gpu(thisParticle, neighbor);
        }
    }
}

__global__ void compute_forces_gpu(particle_t* particles, int num_parts, int* particle_ids, int* prefixSums, int numBoxes1D, double boxSize1D) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    //
    // TODO: check indexing through particle_ids array
    // Access through particle_ids array for coalesced memory access
    // int parts_idx = particle_ids[tid];
    // particle_t& thisParticle = particles[parts_idx];
    //

    particle_t& thisParticle = particles[tid];
    thisParticle.ax = thisParticle.ay = 0;
    int row = findRow(thisParticle, boxSize1D);
    int col = findCol(thisParticle, boxSize1D);

    // TODO: profile loop unrolling
    apply_force_from_neighbor_gpu(row - 1, col - 1, thisParticle, particles, particle_ids, prefixSums, numBoxes1D, boxSize1D); // Up Left
    apply_force_from_neighbor_gpu(row - 1, col, thisParticle, particles, particle_ids, prefixSums, numBoxes1D, boxSize1D);     // Up
    apply_force_from_neighbor_gpu(row - 1, col + 1, thisParticle, particles, particle_ids, prefixSums, numBoxes1D, boxSize1D); // Up Right
    apply_force_from_neighbor_gpu(row, col - 1, thisParticle, particles, particle_ids, prefixSums, numBoxes1D, boxSize1D);     // Left
    apply_force_from_neighbor_gpu(row, col + 1, thisParticle, particles, particle_ids, prefixSums, numBoxes1D, boxSize1D);     // Right
    apply_force_from_neighbor_gpu(row + 1, col - 1, thisParticle, particles, particle_ids, prefixSums, numBoxes1D, boxSize1D); // Down Left
    apply_force_from_neighbor_gpu(row + 1, col, thisParticle, particles, particle_ids, prefixSums, numBoxes1D, boxSize1D);     // Down
    apply_force_from_neighbor_gpu(row + 1, col + 1, thisParticle, particles, particle_ids, prefixSums, numBoxes1D, boxSize1D); // Down Right
    apply_force_from_neighbor_gpu(row, col, thisParticle, particles, particle_ids, prefixSums, numBoxes1D, boxSize1D);         // Self
}

__global__ void move_gpu(particle_t* particles, int num_parts, double size) {

    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    particle_t* p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x += p->vx * dt;
    p->y += p->vy * dt;

    //
    //  bounce from walls
    //
    while (p->x < 0 || p->x > size) {
        p->x = p->x < 0 ? -(p->x) : 2 * size - p->x;
        p->vx = -(p->vx);
    }
    while (p->y < 0 || p->y > size) {
        p->y = p->y < 0 ? -(p->y) : 2 * size - p->y;
        p->vy = -(p->vy);
    }
}

// Iterates through parts and increments boxCounts
__global__ void countParticlesPerBox(particle_t* gpu_parts, int num_parts, int* gpu_boxCounts, int numBoxes1D, double boxSize1D) {
    // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_parts)
        return;

    int boxIndex = findBox(gpu_parts[tid], numBoxes1D, boxSize1D);
    // printf("cur parts idx: %i. boxIndex: %i. Coords: (%f, %f)\n", tid, boxIndex, gpu_parts[tid].x, gpu_parts[tid].y);
    gpu_boxCounts[boxIndex]++;
}

// Iterates through boxCounts and computes a prefixSum
void computePrefixSum() {
    int prefixSum = 0;
    for (int boxIndex = 0; boxIndex <= totalBoxes; ++boxIndex) {
        prefixSums[boxIndex] = prefixSum;
        prefixSum += boxCounts[boxIndex];
        // printf("%i\n", boxCounts[boxIndex]);
    }
}
// Organizes parts by box, in particle_id array
// Uses prefixSum and a reset boxCounts to compute where in particle_id the particle should be inserted 
void populateParticleID(particle_t* parts, int num_parts) {
    memset(boxCounts, 0, boxesMemSize);
    for (int i = 0; i < num_parts; ++i) {
        int boxIndex = findBox(parts[i], numBoxes1D, boxSize1D);
        int pos = prefixSums[boxIndex] + boxCounts[boxIndex];
        particle_ids[pos] = i;
        boxCounts[boxIndex]++;
    }
}

void printAssignmentStats(particle_t* parts) {
    int numEmpty = 0;
    int numFilled = 0;
    int partCount = 0;
    for (int i = 0; i < totalBoxes; ++i) {
        // prefixSums[i] = (boxCounts[i] > 0) ? prefixSums[i] : -1;
        if (boxCounts[i] == 0) {
            printf("Check for empty box %i. prefixSums[%i]: %i\n", i, i, prefixSums[i]);
            numEmpty += 1;
        }
        else {
            numFilled += 1;
            partCount += boxCounts[i];
            printf("Particle id: %i with coords: (%f, %f) in box %i. findBox_output: %i.\n",
            particle_ids[prefixSums[i]], parts[particle_ids[prefixSums[i]]].x, parts[particle_ids[prefixSums[i]]].y, i, findBox(parts[particle_ids[prefixSums[i]]], numBoxes1D, boxSize1D));
        }
    }
    printf("prefixSums[0]: %i. prefixSums[totalBoxes]: %i\n", prefixSums[0], prefixSums[totalBoxes]);
    printf("Num empty boxes: %i. Num boxes w/ particles: %i. Num particles: %i. Average particles per box: %f\n", 
        numEmpty, numFilled, partCount, (double)(partCount / numFilled));
}

// Initializes the particle_id and prefixSums arrays, on GPU
void assignToBoxes(particle_t* parts, int num_parts, int* gpu_boxCounts) {
    // setbuf(stdout, NULL);

    // Copy from parts (gpu_parts) to cpu_parts
    particle_t* cpu_parts = new particle_t[num_parts];
    hipMemcpy(cpu_parts, parts, particle_idMemSize, hipMemcpyDeviceToHost);    

    // First pass: count particles in each box. Reset box counts from past iteration
    hipMemset(gpu_boxCounts, 0, boxesMemSize);
    countParticlesPerBox<<<blks, NUM_THREADS>>>(parts, num_parts, gpu_boxCounts, numBoxes1D, boxSize1D);


    //
    // TEST countParticlesPerBox
    // Use thrust to calculate the sum of all values in gpu_boxCounts
    thrust::device_ptr<int> dev_ptr(gpu_boxCounts);
    int totalParticles = thrust::reduce(dev_ptr, dev_ptr + totalBoxes, 0, thrust::plus<int>());
    printf("Sum of gpu_boxCounts: %d\n", totalParticles);
    

    // Wait for all threads to finish. Then copy gpu_boxCounts to CPU, use for computePrefixSum
    hipDeviceSynchronize();
    hipMemcpy(boxCounts, gpu_boxCounts, boxesMemSize, hipMemcpyDeviceToHost);

    // TEST cpu boxCounts sum
    int totalParticlesCPU = std::accumulate(boxCounts, boxCounts + totalBoxes, 0);
    printf("Sum of cpu boxCounts: %d\n", totalParticlesCPU);

    // Compute starting index for each box in particle_idx from boxCounts
    computePrefixSum();

    populateParticleID(cpu_parts, num_parts);

    // printAssignmentStats(cpu_parts);
}

// Copies data from CPU particle_id and prefixSums to mirrored arrs on GPU
void copyArraysToGPU() {
    hipMemcpy(gpu_particle_ids, particle_ids, particle_idMemSize, hipMemcpyHostToDevice);
    hipMemcpy(gpu_prefixSums, prefixSums, prefixMemSize, hipMemcpyHostToDevice);
}

void init_simulation(particle_t* parts, int num_parts, double size) {
    // You can use this space to initialize data objects that you may need
    // This function will be called once before the algorithm begins
    // parts live in GPU memory
    // Do not do any particle simulation here
    setbuf(stdout, NULL);

    // Assign global variables
    blks = (num_parts + NUM_THREADS - 1) / NUM_THREADS;
    numBoxes1D = (int) ceil(size / boxSize1D);
    totalBoxes = numBoxes1D * numBoxes1D;
    boxesMemSize = totalBoxes * sizeof(int);
    prefixMemSize = (totalBoxes + 1) * sizeof(int);
    particle_idMemSize = num_parts * sizeof(int);

    // Allocate memory for CPU-side arrays
    boxCounts = new int[totalBoxes]();
    prefixSums = new int[totalBoxes + 1];
    particle_ids = new int[num_parts];

    // Allocate memory for GPU-side arrays and copy from CPU-side arrays
    hipMalloc((void**)&gpu_boxCounts, boxesMemSize);
    hipMemset(gpu_boxCounts, 0, boxesMemSize);
    hipMalloc((void**)&gpu_prefixSums, prefixMemSize);
    hipMalloc((void**)&gpu_particle_ids, num_parts * sizeof(int));

    // printf("Numboxes1d: %i. totalBoxes: %i\n", numBoxes1D, totalBoxes);
}

void simulate_one_step(particle_t* parts, int num_parts, double size) {
    // parts live in GPU memory
    // Rewrite this function

    // Assign all particles to boxes
    assignToBoxes(parts, num_parts, gpu_boxCounts);

    // Copy CPU arrays that were updated by assignToBoxes to GPU
    copyArraysToGPU();

    // Compute forces
    compute_forces_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, gpu_particle_ids, gpu_prefixSums, numBoxes1D, boxSize1D);

    // Move particles
    move_gpu<<<blks, NUM_THREADS>>>(parts, num_parts, size);
}